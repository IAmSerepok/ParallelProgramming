#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define EPS (0.001)
#define N (4 * 1)
#define KERNEL (4)


__global__ void integrateBodies(float3 *newPos, float3 *newVel, float3 *oldPos, float3 *oldVel, float dt){
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    float3 pos = oldPos[index];
    float3 f = make_float3(0.0, 0.0, 0.0);

    for(int i = 0; i < N; ++i){
        float3 pi = oldPos[i];
        float3 r;

        r.x = pi.x - pos.x;
        r.y = pi.y - pos.y;
        r.z = pi.z - pos.z;

        float invDist = 1.0 / sqrtf(r.x * r.x + r.y * r.y + r.z * r.z + EPS * EPS);
        float s = invDist * invDist * invDist;

        f.x += r.x * s;
        f.y += r.y * s;
        f.z += r.z * s;
    }

    float3 vel = oldVel[index];

    vel.x += f.x * dt;
    vel.y += f.y * dt;
    vel.z += f.z * dt;

    pos.x += vel.x * dt;
    pos.y += vel.y * dt;
    pos.z += vel.x * dt;

    newPos[index] = pos;
    newVel[index] = vel;
}


void randomInit(float3 *a, int n){
    for(int i = 0; i < n; ++i){
        a[i].x = rand() / (float) RAND_MAX - 0.5;
        a[i].y = rand() / (float) RAND_MAX - 0.5;
        a[i].z = rand() / (float) RAND_MAX - 0.5;
        std::cout << a[i].x << " " << a[i].y << " " << a[i].z << "\n";
    }
    std::cout << "\n";
}


int main(){
    float3 *p = new float3[N];
    float3 *v = new float3[N];

    float3 *pDev[2] = {NULL, NULL};
    float3 *vDev[2] = {NULL, NULL};

    hipEvent_t start, stop;

    int index = 0;

    float gpuTime = 0.0;

    randomInit(p, N);
    randomInit(v, N);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc((void **) &pDev[0], N * sizeof(float3));
    hipMalloc((void **) &pDev[1], N * sizeof(float3));
    hipMalloc((void **) &vDev[0], N * sizeof(float3));
    hipMalloc((void **) &vDev[1], N * sizeof(float3));

    hipMemcpy(pDev[0], p, N * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(vDev[0], v, N * sizeof(float3), hipMemcpyHostToDevice);

    for(int i = 0; i < 4; i++, index ^= 1)
        integrateBodies <<<dim3(N / KERNEL), dim3(KERNEL)>>> (pDev[index^1], vDev[index^1], pDev[index], vDev[index], 0.01);

    hipMemcpy(p, pDev[index^1], N * sizeof(float3), hipMemcpyDeviceToHost);
    hipMemcpy(v, vDev[index^1], N * sizeof(float3), hipMemcpyDeviceToHost);

    hipFree(pDev[0]);
    hipFree(pDev[1]);
    hipFree(vDev[0]);
    hipFree(vDev[1]);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpuTime, start, stop);

    printf("Elapsed time: %.2f\n", gpuTime);

    delete p, 
    delete v;

    return 0;
}
