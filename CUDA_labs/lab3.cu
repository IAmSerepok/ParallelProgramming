#include <iostream>
#include <hip/hip_runtime.h>

#define N 10000

__global__ void VecAdd(float* A, float* B, float* C)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

int main()
{
    float *A = new float[N];
    float *B = new float[N];
    float *C = new float[N];

    for (int i = 0; i < N; ++i) {
        A[i] = i;
        B[i] = i;
    }
   
    float *dev_A, *dev_B, *dev_C;
    hipMalloc((void**)&dev_A, sizeof(float) * N);
    hipMalloc((void**)&dev_B, sizeof(float) * N);
    hipMalloc((void**)&dev_C, sizeof(float) * N);

    hipMemcpy(dev_A, A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, sizeof(float) * N, hipMemcpyHostToDevice);

    // Создание событий для измерения времени
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Запись времени начала
    hipEventRecord(start, 0);

    // Запуск ядра
    VecAdd<<<(N + 255) / 256, 256>>>(dev_A, dev_B, dev_C);
    hipDeviceSynchronize();

    // Запись времени окончания
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    
    // Получение времени выполнения
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(C, dev_C, sizeof(float) * N, hipMemcpyDeviceToHost);

    std::cout << "C[1000]: " << C[1000] << " C[1001]: " << C[1001] << std::endl;
    std::cout << "Время выполнения на GPU: " << milliseconds << " мс" << std::endl;

    delete[] A;
    delete[] B;
    delete[] C;

    hipFree(dev_A);
    hipFree(dev_B);
    hipFree(dev_C);

    // Освобождение событий
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
